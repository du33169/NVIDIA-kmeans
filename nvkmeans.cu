//nvkmeans

#include <thrust/device_vector.h>
#include "kmeans.h"

#include <iostream>
extern "C"
{
	using std::cout;
	using std::endl;
 __attribute__((visibility ("default"))) int fit(
	//input
	int maxIter, int n, int d, int k, double *dataset, 
	//input initial center, output final center
	double *centers,
	//output
	int *labels

	)
{
	cout<<"[nvkmeans] n="<<n<<" d="<<d<<" k="<<k<<endl;	
	thrust::device_vector<double> *data_v[1];
    thrust::device_vector<int> *labels_v[1];
    thrust::device_vector<double> *centroids_v[1];
    thrust::device_vector<double> *distances_v[1];
	cout<<"[nvkmeans]creating device vectors"<<endl;
    data_v[0]      = new thrust::device_vector<double>(dataset, dataset+n*d);
    centroids_v[0] = new thrust::device_vector<double>(centers,centers+k*d);
    
	distances_v[0] = new thrust::device_vector<double>(n);
    labels_v[0]    = new thrust::device_vector<int>(n);

	bool init_from_labels = false;
	double threshold = 1e-6;
	int n_gpu=1;
	cout<<("[nvkmeans]running kmeans")<<endl;
	int iter = kmeans::kmeans(maxIter, n, d, k, data_v, labels_v, centroids_v, distances_v, n_gpu, init_from_labels,threshold);
	cout<<("[nvkmeans]copy back data")<<endl;
	int* labels_dv= thrust::raw_pointer_cast(labels_v[0]->data());
	hipError_t ret=hipMemcpy(labels, labels_dv, n * sizeof(int), hipMemcpyDeviceToHost);
	if(ret!=hipSuccess)
		cout << '[nvkmeans] copy back labels failed.' << endl;
	double *centers_dv = thrust::raw_pointer_cast(centroids_v[0]->data());
	hipError_t ret=hipMemcpy(centers, centers_dv, k*d * sizeof(double), hipMemcpyDeviceToHost);
	if(ret!=hipSuccess)
		cout << '[nvkmeans] copy back centers failed.' << endl;
	double *centers_dv = thrust::raw_pointer_cast(centroids_v[0]->data());
	cout<<"labels:";
	for(int i=0;i<=20;++i){cout<<(*labels_v[0])[i]<<',';}
	cout<<endl;
	cout<<("[nvkmeans]cleaning")<<endl;
	//clean
	delete (data_v[0]);
	delete(labels_v[0]);
    delete(centroids_v[0]);
    delete(distances_v[0]);
	cout<<("[nvkmeans]done")<<endl;
	return iter;
}

}
