//nvkmeans

#include <thrust/device_vector.h>
#include "kmeans.h"

void random_labels(thrust::device_vector<int>& labels, int n, int k) {
    thrust::host_vector<int> host_labels(n);
    for(int i = 0; i < n; i++) {
        host_labels[i] = rand() % k;
    }
    labels = host_labels;
}

extern "C"
{
 __attribute__((visibility ("default"))) int fit(
	//input
	int maxIter, int n, int d, int k, double *dataset, 
	
	//output
	double *centers,
	int *labels

	)
{
	thrust::device_vector<double> *data_v[1];
    thrust::device_vector<int> *labels_v[1];
    thrust::device_vector<double> *centroids_v[1];
    thrust::device_vector<double> *distances_v[1];

    data_v[0]      = new thrust::device_vector<double>(dataset, dataset+n*d);
    centroids_v[0] = new thrust::device_vector<double>(k*d);
    
	distances_v[0] = new thrust::device_vector<double>(n);
    labels_v[0]    = new thrust::device_vector<int>(n);

	random_labels(*labels_v[0], n, k);
	bool init_from_labels = true;
	double threshold = 1e-4;
	int n_gpu=1;
	int iter = kmeans::kmeans(maxIter, n, d, k, data_v, labels_v, centroids_v, distances_v, n_gpu, init_from_labels,threshold);

	int* labels_dv= thrust::raw_pointer_cast(labels_v[0]->data());
	hipMemcpy(labels, labels_dv, n * sizeof(int), hipMemcpyDeviceToHost);
	double* centers_dv= thrust::raw_pointer_cast(centroids_v[0]->data());
	hipMemcpy(centers, centers_dv, k*d * sizeof(int), hipMemcpyDeviceToHost);
	//clean
	delete (data_v[0]);
	delete(labels_v[0]);
    delete(centroids_v[0]);
    delete(distances_v[0]);
	return iter;
}

}