//nvkmeans

#include <thrust/device_vector.h>
#include "kmeans.h"

void random_labels(thrust::device_vector<int>& labels, int n, int k) {
    thrust::host_vector<int> host_labels(n);
    for(int i = 0; i < n; i++) {
        host_labels[i] = rand() % k;
    }
    labels = host_labels;
}

#include <cstdio>
extern "C"
{
 __attribute__((visibility ("default"))) int fit(
	//input
	int maxIter, int n, int d, int k, double *dataset, 
	//input initial center, output final center
	double *centers,
	//output
	int *labels

	)
{
	thrust::device_vector<double> *data_v[1];
    thrust::device_vector<int> *labels_v[1];
    thrust::device_vector<double> *centroids_v[1];
    thrust::device_vector<double> *distances_v[1];
	printf("[nvkmeans]creating device vectors\n");
    data_v[0]      = new thrust::device_vector<double>(dataset, dataset+n*d);
    centroids_v[0] = new thrust::device_vector<double>(centers,centers+k*d);
    
	distances_v[0] = new thrust::device_vector<double>(n);
    labels_v[0]    = new thrust::device_vector<int>(n);

	bool init_from_labels = false;
	double threshold = 1e-4;
	int n_gpu=1;
	printf("[nvkmeans]running kmeans\n");
	int iter = kmeans::kmeans(maxIter, n, d, k, data_v, labels_v, centroids_v, distances_v, n_gpu, init_from_labels,threshold);
	printf("[nvkmeans]copy back data\n");
	int* labels_dv= thrust::raw_pointer_cast(labels_v[0]->data());
	hipMemcpy(labels, labels_dv, n * sizeof(int), hipMemcpyDeviceToHost);
	double* centers_dv= thrust::raw_pointer_cast(centroids_v[0]->data());
	hipMemcpy(centers, centers_dv, k*d * sizeof(double), hipMemcpyDeviceToHost);
	printf("[nvkmeans]cleaning\n");
	//clean
	delete (data_v[0]);
	delete(labels_v[0]);
    delete(centroids_v[0]);
    delete(distances_v[0]);
	printf("[nvkmeans]done\n");
	return iter;
}

}